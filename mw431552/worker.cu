#include "hip/hip_runtime.h"
#include "algorithm.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <iostream>
#include <fstream>
#include <vector>
#include <string>

#define THREADS_PER_BLOCK 256

__global__ void pheromoneUpdateKernel(
    float alpha,
    float beta,
    float evaporation_rate,
    float Q,
    float *pheromone,
    int *tours,
    int n_cities,
    int m,
    float *choice_info,
    float *distances,
    float *tour_lengths
) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= n_cities * n_cities) return;

    int local_X = tid % n_cities;
    int local_Y = tid / n_cities;

    pheromone[tid] *= (1.0f - evaporation_rate);

    float pheromone_update_value = 0.0f;

    for (int i = 0; i < m; i++) {
        int offset = i * n_cities;
        for (int j = 0; j < n_cities - 1; j++) {
            if ((tours[offset + j] == local_X && tours[offset + j + 1] == local_Y) ||
                (tours[offset + j] == local_Y && tours[offset + j + 1] == local_X))  {
                pheromone_update_value += Q / tour_lengths[i];
            }
        }
        if ((tours[offset + n_cities - 1] == local_X && tours[offset] == local_Y) ||
            (tours[offset + n_cities - 1] == local_Y && tours[offset] == local_X)) {
            pheromone_update_value += Q / tour_lengths[i];
        }
    }

    pheromone[tid] += pheromone_update_value;

    if (distances[local_X * n_cities + local_Y] > 0.0f) {
        float tau = __powf(pheromone[tid], alpha);
        float eta = __powf(1.0f / distances[local_X * n_cities + local_Y], beta);
        choice_info[tid] = tau * eta;
    } else {
        choice_info[tid] = 0.0f;
    }
}

__global__ void init_rng(hiprandState* states, unsigned long seed) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init(seed, idx, 0, &states[idx]);
}

__global__ void workerAntKernel(
    int m, int n_cities,
    int* tours,
    float* choice_info,
    float* selection_prob_all,
    bool* visited,
    float* tour_lengths,
    float* distances,
    hiprandState* states
) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= m) return;

    int offset = tid * n_cities;
    hiprandState localState = states[tid];

    for (int i = 0; i < n_cities; i++) {
        visited[offset + i] = false;
    }

    int step = 0;
    int current_city = 0;
    tours[offset + step] = current_city;
    visited[offset + current_city] = true;
    float tour_len = 0.0f;

    step++;

    while (step < n_cities) {
        float sum_probs = 0.0f;
        for (int j = 0; j < n_cities; j++) {
            if (visited[offset + j]) {
                selection_prob_all[offset + j] = 0.0f;
            } else {
                float prob = choice_info[current_city * n_cities + j];
                selection_prob_all[offset + j] = prob;
                sum_probs += prob;
            }
        }

        if (sum_probs == 0.0f) break;

        float r = hiprand_uniform(&localState) * sum_probs;
        float cumulative_prob = 0.0f;
        int next_city = -1;

        for (int j = 0; j < n_cities; j++) {
            cumulative_prob += selection_prob_all[offset + j];
            if (cumulative_prob >= r && cumulative_prob > 0) {
                next_city = j;
                break;
            }
        }

        tours[offset + step] = next_city;
        visited[offset + next_city] = true;
        tour_len += distances[current_city * n_cities + next_city];

        current_city = next_city;
        step++;
    }

    tour_len += distances[current_city * n_cities + tours[offset]];
    tour_lengths[tid] = tour_len;

    states[tid] = localState;
}

std::string prepare_output_path(const std::string& output_file) {
    if (output_file.find('/') == std::string::npos && output_file.find('\\') == std::string::npos) {
        return "./" + output_file;
    } else {
        return output_file;
    }
}

void worker(const std::vector<std::vector<float>>& graph, int num_iter, double alpha, double beta, double evaporate, int seed, std::string output_file) {
    std::cout << "Running WORKER algorithm with CUDA...\n";

    int n_cities = graph.size();
    int m = n_cities; // number of ants = number of cities
    float Q = 100.0f;

    size_t matrix_size = n_cities * n_cities * sizeof(float);
    size_t array_size = m * n_cities * sizeof(int);
    size_t bool_array_size = m * n_cities * sizeof(bool);
    size_t float_array_size = m * n_cities * sizeof(float);
    size_t tour_lengths_size = m * sizeof(float);

    // Host distances matrix
    std::vector<float> distances_host(n_cities * n_cities);
    for (int i = 0; i < n_cities; ++i) {
        for (int j = 0; j < n_cities; ++j) {
            distances_host[i * n_cities + j] = graph[i][j];
        }
    }

    // Device memory
    float *d_pheromone, *d_choice_info, *d_distances, *d_selection_prob_all, *d_tour_lengths;
    int *d_tours;
    bool *d_visited;
    hiprandState* d_states;

    hipMalloc(&d_pheromone, matrix_size);
    hipMalloc(&d_choice_info, matrix_size);
    hipMalloc(&d_distances, matrix_size);
    hipMalloc(&d_tours, array_size);
    hipMalloc(&d_selection_prob_all, float_array_size);
    hipMalloc(&d_visited, bool_array_size);
    hipMalloc(&d_tour_lengths, tour_lengths_size);
    hipMalloc(&d_states, m * sizeof(hiprandState));

    hipMemcpy(d_distances, distances_host.data(), matrix_size, hipMemcpyHostToDevice);
    hipMemset(d_pheromone, 0, matrix_size);

    int blocks = (m + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    int blocks_matrix = (n_cities * n_cities + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

    init_rng<<<blocks, THREADS_PER_BLOCK>>>(d_states, seed);
    hipDeviceSynchronize();

    for (int iter = 0; iter < num_iter; ++iter) {
        workerAntKernel<<<blocks, THREADS_PER_BLOCK>>>(m, n_cities, d_tours, d_choice_info, d_selection_prob_all, d_visited, d_tour_lengths, d_distances, d_states);
        hipDeviceSynchronize();

        pheromoneUpdateKernel<<<blocks_matrix, THREADS_PER_BLOCK>>>(
            static_cast<float>(alpha),
            static_cast<float>(beta),
            static_cast<float>(evaporate),
            Q,
            d_pheromone,
            d_tours,
            n_cities,
            m,
            d_choice_info,
            d_distances,
            d_tour_lengths
        );
        hipDeviceSynchronize();
    }

    std::vector<float> tour_lengths_host(m);
    hipMemcpy(tour_lengths_host.data(), d_tour_lengths, tour_lengths_size, hipMemcpyDeviceToHost);

    float best = 1e9;
    for (int i = 0; i < m; ++i) {
        if (tour_lengths_host[i] < best) {
            best = tour_lengths_host[i];
        }
    }

    std::string full_output_path = prepare_output_path(output_file);
    std::ofstream ofs(full_output_path);
    if (!ofs.is_open()) {
        std::cerr << "Error opening output file: " << full_output_path << std::endl;
        return;
    }

    ofs << best << "\n";
    ofs.close();

    hipFree(d_pheromone);
    hipFree(d_choice_info);
    hipFree(d_distances);
    hipFree(d_tours);
    hipFree(d_selection_prob_all);
    hipFree(d_visited);
    hipFree(d_tour_lengths);
    hipFree(d_states);

    std::cout << "Best tour length: " << best << std::endl;
}
