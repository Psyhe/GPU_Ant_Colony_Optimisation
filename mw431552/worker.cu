#include "hip/hip_runtime.h"
#include "algorithm.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <iostream>
#include <fstream>
#include <vector>
#include <string>
#include <algorithm>
#include <iomanip> // for better formatting

#define N_MAX_THREADS_PER_BLOCK 1024


__global__ void pheromoneUpdateKernel(
    float alpha,
    float beta,
    float evaporation_rate,
    float Q,
    float *pheromone,
    int *tours,
    int n_cities,
    int m,
    float *choice_info,
    float *distances,
    float *tour_lengths
) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= n_cities * n_cities) return;

    int local_X = tid % n_cities;
    int local_Y = tid / n_cities;

    pheromone[tid] *= (1.0f - evaporation_rate);

    float pheromone_update_value = 0.0f;

    for (int i = 0; i < m; i++) {
        int offset = i * n_cities;
        for (int j = 0; j < n_cities - 1; j++) {
            if ((tours[offset + j] == local_X && tours[offset + j + 1] == local_Y) ||
                (tours[offset + j] == local_Y && tours[offset + j + 1] == local_X))  {
                pheromone_update_value += Q / tour_lengths[i];
            }
        }
        if ((tours[offset + n_cities - 1] == local_X && tours[offset] == local_Y) ||
            (tours[offset + n_cities - 1] == local_Y && tours[offset] == local_X)) {
            pheromone_update_value += Q / tour_lengths[i];
        }
    }

    pheromone[tid] += pheromone_update_value;

    if (distances[local_X * n_cities + local_Y] > 0.0f) {
        float tau = __powf(pheromone[tid], alpha);
        float eta = __powf(1.0f / distances[local_X * n_cities + local_Y], beta);
        choice_info[tid] = tau * eta;
    } else {
        choice_info[tid] = 0.0f;
    }
}

__global__ void init_rng(hiprandState* states, unsigned long seed) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init(seed, idx, 0, &states[idx]);
}

__global__ void workerAntKernel(
    int m, int n_cities,
    int* tours,
    float* choice_info,
    float* selection_prob_all,
    bool* visited,
    float* tour_lengths,
    float* distances,
    hiprandState* states
) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= m) return;

    int offset = tid * n_cities;
    hiprandState localState = states[tid];

    for (int i = 0; i < n_cities; i++) {
        visited[offset + i] = false;
    }

    int step = 0;
    int current_city = 0;
    tours[offset + step] = current_city;
    visited[offset + current_city] = true;
    float tour_len = 0.0f;

    step++;

    while (step < n_cities) {
        float sum_probs = 0.0f;
        for (int j = 0; j < n_cities; j++) {
            if (visited[offset + j]) {
                selection_prob_all[offset + j] = 0.0f;
            } else {
                float prob = choice_info[current_city * n_cities + j];
                selection_prob_all[offset + j] = prob;
                sum_probs += prob;
            }
        }

        if (sum_probs == 0.0f) break;

        float r = hiprand_uniform(&localState) * sum_probs;
        float cumulative_prob = 0.0f;
        int next_city = -1;

        for (int j = 0; j < n_cities; j++) {
            cumulative_prob += selection_prob_all[offset + j];
            if (cumulative_prob >= r && cumulative_prob > 0) {
                next_city = j;
                break;
            }
        }

        tours[offset + step] = next_city;
        visited[offset + next_city] = true;
        tour_len += distances[current_city * n_cities + next_city];

        current_city = next_city;
        step++;
    }

    tour_len += distances[current_city * n_cities + tours[offset]];
    tour_lengths[tid] = tour_len;

    states[tid] = localState;
}

std::string prepare_output_path(const std::string& output_file) {
    if (output_file.find('/') == std::string::npos && output_file.find('\\') == std::string::npos) {
        return "./" + output_file;
    } else {
        return output_file;
    }
}

void worker(const std::vector<std::vector<float>>& graph, int num_iter, float alpha, float beta, float evaporate, int seed, std::string output_file) {
    std::cout << "Running WORKER algorithm with CUDA...\n";

    int n_cities = graph.size();
    int m = n_cities; // number of ants = number of cities
    float Q = 1.0f;

    size_t matrix_size = n_cities * n_cities * sizeof(float);
    size_t array_size = m * n_cities * sizeof(int);
    size_t bool_array_size = m * n_cities * sizeof(bool);
    size_t float_array_size = m * n_cities * sizeof(float);
    size_t tour_lengths_size = m * sizeof(float);

    // Host distances matrix
    std::vector<float> distances_host(n_cities * n_cities);
    for (int i = 0; i < n_cities; ++i) {
        for (int j = 0; j < n_cities; ++j) {
            distances_host[i * n_cities + j] = graph[i][j];
        }
    }

    

    // Device memory
    float *d_pheromone, *d_choice_info, *d_distances, *d_selection_prob_all, *d_tour_lengths;
    int *d_tours;
    bool *d_visited;
    hiprandState* d_states;

    hipMalloc(&d_pheromone, matrix_size);
    hipMalloc(&d_choice_info, matrix_size);
    hipMalloc(&d_distances, matrix_size);
    hipMalloc(&d_tours, array_size);
    hipMalloc(&d_selection_prob_all, float_array_size);
    hipMalloc(&d_visited, bool_array_size);
    hipMalloc(&d_tour_lengths, tour_lengths_size);
    hipMalloc(&d_states, m * sizeof(hiprandState));

    hipMemcpy(d_distances, distances_host.data(), matrix_size, hipMemcpyHostToDevice);
    std::vector<float> initial_pheromone(n_cities * n_cities, 1.0f);
    hipMemcpy(d_pheromone, initial_pheromone.data(), matrix_size, hipMemcpyHostToDevice);

    int n_ants = n_cities;

    int thread_worker_count = min(N_MAX_THREADS_PER_BLOCK, n_ants);
    int blocks_worker = (n_ants / thread_worker_count) + 1;

    // int threads_count = n_cities;
    // int blocks = (m + threads_count - 1) / threads_count; // enough blocks for all ants

    init_rng<<<blocks_worker, thread_worker_count>>>(d_states, seed);
    hipDeviceSynchronize();

    int all_threads_pheromone = n_ants * n_ants;
    int threads_pheromone = min(N_MAX_THREADS_PER_BLOCK, all_threads_pheromone);
    int blocks_pheromone = (all_threads_pheromone / threads_pheromone) + 1;

    // Host buffers to fetch data back from GPU
    std::vector<int> tours_host(m * n_cities);
    std::vector<float> choice_info_host(n_cities * n_cities);
    std::vector<float> tour_lengths_host(m);

    for (int iter = 0; iter < num_iter; ++iter) {
        // std::cout << "\n=== Iteration " << iter + 1 << " ===\n";

        workerAntKernel<<<blocks_worker, thread_worker_count>>>(m, n_cities, d_tours, d_choice_info, d_selection_prob_all, d_visited, d_tour_lengths, d_distances, d_states);
        hipDeviceSynchronize();

        pheromoneUpdateKernel<<<blocks_pheromone, threads_pheromone>>>(
            alpha, 
            beta,
            evaporate,
            Q,
            d_pheromone,
            d_tours,
            n_cities,
            m,
            d_choice_info,
            d_distances,
            d_tour_lengths
        );
        hipDeviceSynchronize();

        // Copy back tours and lengths
        hipMemcpy(tours_host.data(), d_tours, array_size, hipMemcpyDeviceToHost);
        hipMemcpy(tour_lengths_host.data(), d_tour_lengths, tour_lengths_size, hipMemcpyDeviceToHost);
        hipMemcpy(choice_info_host.data(), d_choice_info, matrix_size, hipMemcpyDeviceToHost);
        hipMemcpy(initial_pheromone.data(), d_pheromone, matrix_size, hipMemcpyDeviceToHost);

        // // Print tours
        // for (int ant = 0; ant < m; ++ant) {
        //     std::cout << "Ant " << ant << " tour: ";
        //     for (int step = 0; step < n_cities; ++step) {
        //         std::cout << tours_host[ant * n_cities + step] << " ";
        //     }
        //     std::cout << " (length: " << tour_lengths_host[ant] << ")\n";
        // }

        // std::cout << "Pheromone Info Matrix:\n";
        // for (int i = 0; i < n_cities; ++i) {
        //     for (int j = 0; j < n_cities; ++j) {
        //         std::cout << std::fixed << std::setprecision(4) << initial_pheromone[i * n_cities + j] << "\t";
        //     }
        //     std::cout << "\n";
        // }

        // // Print choice_info matrix
        // std::cout << "Choice Info Matrix:\n";
        // for (int i = 0; i < n_cities; ++i) {
        //     for (int j = 0; j < n_cities; ++j) {
        //         std::cout << std::fixed << std::setprecision(4) << choice_info_host[i * n_cities + j] << "\t";
        //     }
        //     std::cout << "\n";
        // }
    }

    float best = 1e9;
    int id = 0;
    for (int i = 0; i < m; ++i) {
        if (tour_lengths_host[i] < best) {
            best = tour_lengths_host[i];
            id = i;
        }
    }

    hipFree(d_pheromone);
    hipFree(d_choice_info);
    hipFree(d_distances);
    hipFree(d_tours);
    hipFree(d_selection_prob_all);
    hipFree(d_visited);
    hipFree(d_tour_lengths);
    hipFree(d_states);

    std::cout << "\nBest tour length: " << best << std::endl;

    for (int i = 0; i < m; i++) {
        std::cout << tours_host[i] << " ";
    }
}