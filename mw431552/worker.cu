#include "hip/hip_runtime.h"
#include "algorithm.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <iostream>
#include <fstream>
#include <vector>
#include <string>
#include <algorithm>
#include <iomanip> // for better formatting
#include <fstream>
#include <chrono>
#include "utils.h"

#define N_MAX_THREADS_PER_BLOCK 1024

// __global__ void workerAntKernel(
//     int m, int n_cities,
//     int* tours,
//     float* choice_info,
//     float* selection_prob_all,
//     bool* visited,
//     float* tour_lengths,
//     float* distances,
//     hiprandState* states
// ) {
//     int tid = blockIdx.x * blockDim.x + threadIdx.x;
//     if (tid >= m) return;

//     int offset = tid * n_cities;
//     hiprandState localState = states[tid];

//     for (int i = 0; i < n_cities; i++) {
//         visited[offset + i] = false;
//     }

//     int step = 0;
//     int current_city = 0;
//     tours[offset + step] = current_city;
//     visited[offset + current_city] = true;
//     float tour_len = 0.0f;

//     step++;

//     while (step < n_cities) {
//         float sum_probs = 0.0f;
//         for (int j = 0; j < n_cities; j++) {
//             if (visited[offset + j]) {
//                 selection_prob_all[offset + j] = 0.0f;
//             } else {
//                 float prob = choice_info[current_city * n_cities + j];
//                 selection_prob_all[offset + j] = prob;
//                 sum_probs += prob;
//             }
//         }

//         if (sum_probs == 0.0f) break;

//         float r = hiprand_uniform(&localState) * sum_probs;
//         float cumulative_prob = 0.0f;
//         int next_city = -1;

//         for (int j = 0; j < n_cities; j++) {
//             cumulative_prob += selection_prob_all[offset + j];
//             if (cumulative_prob >= r) {
//                 next_city = j;
//                 break;
//             }
//         }

//         tours[offset + step] = next_city;
//         visited[offset + next_city] = true;
//         tour_len += distances[current_city * n_cities + next_city];

//         current_city = next_city;
//         step++;
//     }

//     tour_len += distances[current_city * n_cities + tours[offset]];
//     tour_lengths[tid] = tour_len;

//     states[tid] = localState;
// }

// void worker(const std::vector<std::vector<float>>& graph, int num_iter, float alpha, float beta, float evaporate, int seed, std::string output_file) {
//     std::cout << "Running WORKER algorithm with CUDA...\n";

//     hipEvent_t start_total, end_total;
//     hipEventCreate(&start_total);
//     hipEventCreate(&end_total);
//     hipEventRecord(start_total);

//     float total_kernel = 0.0f;
//     float total_pheromone = 0.0f;

//     int n_cities = graph.size();
//     int m = n_cities;
//     float Q = 1.0f;

//     size_t matrix_size = n_cities * n_cities * sizeof(float);
//     size_t array_size = m * n_cities * sizeof(int);
//     size_t bool_array_size = m * n_cities * sizeof(bool);
//     size_t float_array_size = m * n_cities * sizeof(float);
//     size_t tour_lengths_size = m * sizeof(float);

//     std::vector<float> distances_host(n_cities * n_cities);
//     for (int i = 0; i < n_cities; ++i) {
//         for (int j = 0; j < n_cities; ++j) {
//             distances_host[i * n_cities + j] = graph[i][j];
//         }
//     }

//     float *d_pheromone, *d_choice_info, *d_distances, *d_selection_prob_all, *d_tour_lengths;
//     int *d_tours;
//     bool *d_visited;
//     hiprandState* d_states;

//     hipMalloc(&d_pheromone, matrix_size);
//     hipMalloc(&d_choice_info, matrix_size);
//     hipMalloc(&d_distances, matrix_size);
//     hipMalloc(&d_tours, array_size);
//     hipMalloc(&d_selection_prob_all, float_array_size);
//     hipMalloc(&d_visited, bool_array_size);
//     hipMalloc(&d_tour_lengths, tour_lengths_size);
//     hipMalloc(&d_states, m * sizeof(hiprandState));

//     hipMemcpy(d_distances, distances_host.data(), matrix_size, hipMemcpyHostToDevice);
//     std::vector<float> initial_pheromone(n_cities * n_cities, 1.0f);
//     hipMemcpy(d_pheromone, initial_pheromone.data(), matrix_size, hipMemcpyHostToDevice);

//     int thread_worker_count = std::min(N_MAX_THREADS_PER_BLOCK, m);
//     int blocks_worker = (m + thread_worker_count - 1) / thread_worker_count;

//     init_rng<<<blocks_worker, thread_worker_count>>>(d_states, seed);
//     hipDeviceSynchronize();

//     int all_threads_pheromone = m * n_cities;
//     int threads_pheromone = std::min(N_MAX_THREADS_PER_BLOCK, all_threads_pheromone);
//     int blocks_pheromone = (all_threads_pheromone + threads_pheromone - 1) / threads_pheromone;

//     std::vector<int> tours_host(m * n_cities);
//     std::vector<float> choice_info_host(n_cities * n_cities);
//     std::vector<float> tour_lengths_host(m);

//     hipEvent_t start_kernel, end_kernel;
//     hipEvent_t start_pheromone, end_pheromone;
//     hipEventCreate(&start_kernel);
//     hipEventCreate(&end_kernel);
//     hipEventCreate(&start_pheromone);
//     hipEventCreate(&end_pheromone);

//     pheromoneUpdateKernel<<<blocks_pheromone, threads_pheromone>>>(
//         alpha, beta, evaporate, Q,
//         d_pheromone, d_tours, n_cities, m,
//         d_choice_info, d_distances, d_tour_lengths
//     );
//     hipDeviceSynchronize();

//     for (int iter = 0; iter < num_iter; ++iter) {
//         hipEventRecord(start_kernel);
//         workerAntKernel<<<blocks_worker, thread_worker_count>>>(
//             m, n_cities, d_tours, d_choice_info, d_selection_prob_all,
//             d_visited, d_tour_lengths, d_distances, d_states
//         );
//         hipDeviceSynchronize();
//         hipEventRecord(end_kernel);
//         hipEventSynchronize(end_kernel);

//         float kernel_time = 0.0f;
//         hipEventElapsedTime(&kernel_time, start_kernel, end_kernel);
//         total_kernel += kernel_time;

//         hipEventRecord(start_pheromone);
//         pheromoneUpdateKernel<<<blocks_pheromone, threads_pheromone>>>(
//             alpha, beta, evaporate, Q,
//             d_pheromone, d_tours, n_cities, m,
//             d_choice_info, d_distances, d_tour_lengths
//         );
//         hipDeviceSynchronize();
//         hipEventRecord(end_pheromone);
//         hipEventSynchronize(end_pheromone);

//         float pheromone_time = 0.0f;
//         hipEventElapsedTime(&pheromone_time, start_pheromone, end_pheromone);
//         total_pheromone += pheromone_time;
//     }

//     hipMemcpy(tours_host.data(), d_tours, array_size, hipMemcpyDeviceToHost);
//     hipMemcpy(tour_lengths_host.data(), d_tour_lengths, tour_lengths_size, hipMemcpyDeviceToHost);
//     hipMemcpy(choice_info_host.data(), d_choice_info, matrix_size, hipMemcpyDeviceToHost);
//     hipMemcpy(initial_pheromone.data(), d_pheromone, matrix_size, hipMemcpyDeviceToHost);

//     float best = 1e9;
//     int best_id = 0;
//     for (int i = 0; i < m; ++i) {
//         if (tour_lengths_host[i] < best) {
//             best = tour_lengths_host[i];
//             best_id = i;
//         }
//     }

//     hipFree(d_pheromone);
//     hipFree(d_choice_info);
//     hipFree(d_distances);
//     hipFree(d_tours);
//     hipFree(d_selection_prob_all);
//     hipFree(d_visited);
//     hipFree(d_tour_lengths);
//     hipFree(d_states);

//     hipEventRecord(end_total);
//     hipEventSynchronize(end_total);

//     float total_time = 0.0f;
//     hipEventElapsedTime(&total_time, start_total, end_total);

//     std::cout << "Total kernel time: " << total_kernel / 1000.0f << " s" << std::endl;
//     std::cout << "Total kernel pheromone time: " << total_pheromone / 1000.0f << " s" << std::endl;
//     std::cout << "Average kernel time: " << total_kernel / num_iter << " ms" << std::endl;
//     std::cout << "Average pheromone kernel time: " << total_pheromone / num_iter << " ms" << std::endl;

//     std::cout << "Total time: " << total_time / 1000.0f << " s" << std::endl;

//     hipEventDestroy(start_total);
//     hipEventDestroy(end_total);
//     hipEventDestroy(start_kernel);
//     hipEventDestroy(end_kernel);
//     hipEventDestroy(start_pheromone);
//     hipEventDestroy(end_pheromone);

//     std::string output_path = prepare_output_path(output_file);
//     std::ofstream out(output_path);

//     if (!out.is_open()) {
//         std::cerr << "Failed to open output file: " << output_path << std::endl;
//         return;
//     }

//     std::cout << "\nBest tour length: " << best << std::endl;
//     out << "Best tour length: " << best << std::endl;

//     for (int step = 0; step < n_cities; ++step) {
//         std::cout << tours_host[best_id * n_cities + step] << " ";
//         out << tours_host[best_id * n_cities + step] + 1 << " ";
//     }
//     std::cout << std::endl;
//     out << std::endl;

//     out.close();
// }



// Rewritten workerAntKernel using bit-packed visited array
__global__ void workerAntKernel(
    int m, int n_cities,
    int* tours,
    float* choice_info,
    float* selection_prob_all,
    bool* visited_global, // unused now
    float* tour_lengths,
    float* distances,
    hiprandState* states
) {
    extern __shared__ unsigned int shared_visited[];
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= m) return;

    const int ints_per_ant = (n_cities + 31) / 32;
    unsigned int* my_visited = &shared_visited[threadIdx.x * ints_per_ant];
    int offset = tid * n_cities;

    hiprandState localState = states[tid];

    // Initialize visited bits
    for (int i = 0; i < ints_per_ant; i++) {
        my_visited[i] = 0;
    }

    int step = 0;
    int current_city = 0;
    tours[offset + step] = current_city;

    // Mark starting city visited
    {
        int idx = current_city / 32;
        int bit = current_city % 32;
        my_visited[idx] |= (1U << bit);
    }

    float tour_len = 0.0f;
    step++;

    while (step < n_cities) {
        float sum_probs = 0.0f;
        for (int j = 0; j < n_cities; j++) {
            int idx = j / 32;
            int bit = j % 32;
            bool is_visited = (my_visited[idx] >> bit) & 1U;

            if (is_visited) {
                selection_prob_all[offset + j] = 0.0f;
            } else {
                float prob = choice_info[current_city * n_cities + j];
                selection_prob_all[offset + j] = prob;
                sum_probs += prob;
            }
        }

        if (sum_probs == 0.0f) break;

        float r = hiprand_uniform(&localState) * sum_probs;
        float cumulative_prob = 0.0f;
        int next_city = -1;

        for (int j = 0; j < n_cities; j++) {
            cumulative_prob += selection_prob_all[offset + j];
            if (cumulative_prob >= r) {
                next_city = j;
                break;
            }
        }

        tours[offset + step] = next_city;

        // Mark next city visited
        {
            int idx = next_city / 32;
            int bit = next_city % 32;
            my_visited[idx] |= (1U << bit);
        }

        tour_len += distances[current_city * n_cities + next_city];

        current_city = next_city;
        step++;
    }

    tour_len += distances[current_city * n_cities + tours[offset]];
    tour_lengths[tid] = tour_len;

    states[tid] = localState;
}

// Your full worker() function, modified only for shared memory size
void worker_old(const std::vector<std::vector<float>>& graph, int num_iter, float alpha, float beta, float evaporate, int seed, std::string output_file) {
    std::cout << "Running WORKER algorithm with CUDA...\n";

    hipEvent_t start_total, end_total;
    hipEventCreate(&start_total);
    hipEventCreate(&end_total);
    hipEventRecord(start_total);

    float total_kernel = 0.0f;
    float total_pheromone = 0.0f;

    int n_cities = graph.size();
    int m = n_cities;
    float Q = 1.0f;

    size_t matrix_size = n_cities * n_cities * sizeof(float);
    size_t array_size = m * n_cities * sizeof(int);
    size_t bool_array_size = m * n_cities * sizeof(bool);
    size_t float_array_size = m * n_cities * sizeof(float);
    size_t tour_lengths_size = m * sizeof(float);

    std::vector<float> distances_host(n_cities * n_cities);
    for (int i = 0; i < n_cities; ++i) {
        for (int j = 0; j < n_cities; ++j) {
            distances_host[i * n_cities + j] = graph[i][j];
        }
    }

    float *d_pheromone, *d_choice_info, *d_distances, *d_selection_prob_all, *d_tour_lengths;
    int *d_tours;
    bool *d_visited;
    hiprandState* d_states;

    hipMalloc(&d_pheromone, matrix_size);
    hipMalloc(&d_choice_info, matrix_size);
    hipMalloc(&d_distances, matrix_size);
    hipMalloc(&d_tours, array_size);
    hipMalloc(&d_selection_prob_all, float_array_size);
    hipMalloc(&d_visited, bool_array_size); // Still needed to match kernel signature
    hipMalloc(&d_tour_lengths, tour_lengths_size);
    hipMalloc(&d_states, m * sizeof(hiprandState));

    hipMemcpy(d_distances, distances_host.data(), matrix_size, hipMemcpyHostToDevice);
    std::vector<float> initial_pheromone(n_cities * n_cities, 1.0f);
    hipMemcpy(d_pheromone, initial_pheromone.data(), matrix_size, hipMemcpyHostToDevice);

    int thread_worker_count = std::min(N_MAX_THREADS_PER_BLOCK, m);
    int blocks_worker = (m + thread_worker_count - 1) / thread_worker_count;

    init_rng<<<blocks_worker, thread_worker_count>>>(d_states, seed);
    hipDeviceSynchronize();

    int all_threads_pheromone = m * n_cities;
    int threads_pheromone = std::min(N_MAX_THREADS_PER_BLOCK, all_threads_pheromone);
    int blocks_pheromone = (all_threads_pheromone + threads_pheromone - 1) / threads_pheromone;

    std::vector<int> tours_host(m * n_cities);
    std::vector<float> choice_info_host(n_cities * n_cities);
    std::vector<float> tour_lengths_host(m);

    hipEvent_t start_kernel, end_kernel;
    hipEvent_t start_pheromone, end_pheromone;
    hipEventCreate(&start_kernel);
    hipEventCreate(&end_kernel);
    hipEventCreate(&start_pheromone);
    hipEventCreate(&end_pheromone);

    pheromoneUpdateKernel<<<blocks_pheromone, threads_pheromone>>>(
        alpha, beta, evaporate, Q,
        d_pheromone, d_tours, n_cities, m,
        d_choice_info, d_distances, d_tour_lengths
    );
    hipDeviceSynchronize();

    const int ints_per_ant = (n_cities + 31) / 32;
    const size_t shared_memory_size = thread_worker_count * ints_per_ant * sizeof(unsigned int);

    for (int iter = 0; iter < num_iter; ++iter) {
        hipEventRecord(start_kernel);
        workerAntKernel<<<blocks_worker, thread_worker_count, shared_memory_size>>>(
            m, n_cities, d_tours, d_choice_info, d_selection_prob_all,
            d_visited, d_tour_lengths, d_distances, d_states
        );
        hipDeviceSynchronize();
        hipEventRecord(end_kernel);
        hipEventSynchronize(end_kernel);

        float kernel_time = 0.0f;
        hipEventElapsedTime(&kernel_time, start_kernel, end_kernel);
        total_kernel += kernel_time;

        hipEventRecord(start_pheromone);
        pheromoneUpdateKernel<<<blocks_pheromone, threads_pheromone>>>(
            alpha, beta, evaporate, Q,
            d_pheromone, d_tours, n_cities, m,
            d_choice_info, d_distances, d_tour_lengths
        );
        hipDeviceSynchronize();
        hipEventRecord(end_pheromone);
        hipEventSynchronize(end_pheromone);

        float pheromone_time = 0.0f;
        hipEventElapsedTime(&pheromone_time, start_pheromone, end_pheromone);
        total_pheromone += pheromone_time;
    }

    hipMemcpy(tours_host.data(), d_tours, array_size, hipMemcpyDeviceToHost);
    hipMemcpy(tour_lengths_host.data(), d_tour_lengths, tour_lengths_size, hipMemcpyDeviceToHost);
    hipMemcpy(choice_info_host.data(), d_choice_info, matrix_size, hipMemcpyDeviceToHost);
    hipMemcpy(initial_pheromone.data(), d_pheromone, matrix_size, hipMemcpyDeviceToHost);

    float best = 1e9;
    int best_id = 0;
    for (int i = 0; i < m; ++i) {
        if (tour_lengths_host[i] < best) {
            best = tour_lengths_host[i];
            best_id = i;
        }
    }

    hipFree(d_pheromone);
    hipFree(d_choice_info);
    hipFree(d_distances);
    hipFree(d_tours);
    hipFree(d_selection_prob_all);
    hipFree(d_visited);
    hipFree(d_tour_lengths);
    hipFree(d_states);

    hipEventRecord(end_total);
    hipEventSynchronize(end_total);

    float total_time = 0.0f;
    hipEventElapsedTime(&total_time, start_total, end_total);

    std::cout << "Total kernel time: " << total_kernel / 1000.0f << " s" << std::endl;
    std::cout << "Total kernel pheromone time: " << total_pheromone / 1000.0f << " s" << std::endl;
    std::cout << "Average kernel time: " << total_kernel / num_iter << " ms" << std::endl;
    std::cout << "Average pheromone kernel time: " << total_pheromone / num_iter << " ms" << std::endl;
    std::cout << "Total time: " << total_time / 1000.0f << " s" << std::endl;

    hipEventDestroy(start_total);
    hipEventDestroy(end_total);
    hipEventDestroy(start_pheromone);
    hipEventDestroy(end_pheromone);

    std::string output_path = prepare_output_path(output_file);
    std::ofstream out(output_path);

    if (!out.is_open()) {
        std::cerr << "Failed to open output file: " << output_path << std::endl;
        return;
    }

    std::cout << "\nBest tour length: " << best << std::endl;
    out << "Best tour length: " << best << std::endl;

    for (int step = 0; step < n_cities; ++step) {
        std::cout << tours_host[best_id * n_cities + step] << " ";
        out << tours_host[best_id * n_cities + step] + 1 << " ";
    }
    std::cout << std::endl;
    out << std::endl;

    out.close();
}

void worker(const std::vector<std::vector<float>>& graph, int num_iter, float alpha, float beta, float evaporate, int seed, std::string output_file) {
    std::cout << "Running WORKER algorithm with CUDA GRAPHS...\n";

    hipEvent_t start_total, end_total;
    hipEventCreate(&start_total);
    hipEventCreate(&end_total);
    hipEventRecord(start_total);

    float total_kernel = 0.0f;
    float total_pheromone = 0.0f;

    int n_cities = graph.size();
    int m = n_cities;
    float Q = 1.0f;

    size_t matrix_size = n_cities * n_cities * sizeof(float);
    size_t array_size = m * n_cities * sizeof(int);
    size_t bool_array_size = m * n_cities * sizeof(bool);
    size_t float_array_size = m * n_cities * sizeof(float);
    size_t tour_lengths_size = m * sizeof(float);

    std::vector<float> distances_host(n_cities * n_cities);
    for (int i = 0; i < n_cities; ++i) {
        for (int j = 0; j < n_cities; ++j) {
            distances_host[i * n_cities + j] = graph[i][j];
        }
    }

    float *d_pheromone, *d_choice_info, *d_distances, *d_selection_prob_all, *d_tour_lengths;
    int *d_tours;
    hiprandState* d_states;

    hipMalloc(&d_pheromone, matrix_size);
    hipMalloc(&d_choice_info, matrix_size);
    hipMalloc(&d_distances, matrix_size);
    hipMalloc(&d_tours, array_size);
    hipMalloc(&d_selection_prob_all, float_array_size);
    hipMalloc(&d_tour_lengths, tour_lengths_size);
    hipMalloc(&d_states, m * sizeof(hiprandState));

    hipMemcpy(d_distances, distances_host.data(), matrix_size, hipMemcpyHostToDevice);
    std::vector<float> initial_pheromone(n_cities * n_cities, 1.0f);
    hipMemcpy(d_pheromone, initial_pheromone.data(), matrix_size, hipMemcpyHostToDevice);

    int thread_worker_count = std::min(N_MAX_THREADS_PER_BLOCK, m);
    int blocks_worker = (m + thread_worker_count - 1) / thread_worker_count;

    init_rng<<<blocks_worker, thread_worker_count>>>(d_states, seed);
    hipDeviceSynchronize();

    // Host buffers
    std::vector<int> tours_host(m * n_cities);
    std::vector<float> choice_info_host(n_cities * n_cities);
    std::vector<float> tour_lengths_host(m);

    int all_threads_pheromone = m * n_cities;
    int threads_pheromone = std::min(N_MAX_THREADS_PER_BLOCK, all_threads_pheromone);
    int blocks_pheromone = (all_threads_pheromone + threads_pheromone - 1) / threads_pheromone;

    const int ints_per_ant = (n_cities + 31) / 32;
    const size_t shared_memory_size = thread_worker_count * ints_per_ant * sizeof(unsigned int);

    hipStream_t stream;
    hipStreamCreate(&stream);

    hipGraph_t graph_capture;
    hipGraphExec_t graph_exec;

    // ====== CAPTURE CUDA GRAPH ======
    hipStream_t streamForGraph;
    hipStreamCreate(&streamForGraph);

    hipStreamBeginCapture(streamForGraph, hipStreamCaptureModeGlobal);

    // These two kernel launches are now *inside* the capture
    workerAntKernel<<<blocks_worker, thread_worker_count, shared_memory_size, streamForGraph>>>(
        m, n_cities, d_tours, d_choice_info, d_selection_prob_all,
        d_visited, d_tour_lengths, d_distances, d_states
    );

    pheromoneUpdateKernel<<<blocks_pheromone, threads_pheromone, 0, streamForGraph>>>(
        alpha, beta, evaporate, Q,
        d_pheromone, d_tours, n_cities, m,
        d_choice_info, d_distances, d_tour_lengths
    );

    hipStreamEndCapture(stream, &graph_capture);
    hipGraphInstantiate(&graph_exec, graph_capture, NULL, NULL, 0);

    runGraphIterations(graph_exec, stream, num_iter, total_kernel);

    hipGraphDestroy(graph_capture);
    hipGraphExecDestroy(graph_exec);
    hipStreamDestroy(stream);

    hipMemcpy(tours_host.data(), d_tours, array_size, hipMemcpyDeviceToHost);
    hipMemcpy(tour_lengths_host.data(), d_tour_lengths, tour_lengths_size, hipMemcpyDeviceToHost);
    hipMemcpy(choice_info_host.data(), d_choice_info, matrix_size, hipMemcpyDeviceToHost);
    hipMemcpy(initial_pheromone.data(), d_pheromone, matrix_size, hipMemcpyDeviceToHost);

    float best = 1e9;
    int best_id = 0;
    for (int i = 0; i < m; ++i) {
        if (tour_lengths_host[i] < best) {
            best = tour_lengths_host[i];
            best_id = i;
        }
    }

    hipFree(d_pheromone);
    hipFree(d_choice_info);
    hipFree(d_distances);
    hipFree(d_tours);
    hipFree(d_selection_prob_all);
    hipFree(d_tour_lengths);
    hipFree(d_states);

    hipEventRecord(end_total);
    hipEventSynchronize(end_total);

    float total_time = 0.0f;
    hipEventElapsedTime(&total_time, start_total, end_total);

    generate_output(total_kernel, num_iter, total_time_ms, output_file, tours_host, best_id, best);
}
