#include "hip/hip_runtime.h"
#include "algorithm.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <iostream>
#include <fstream>
#include <vector>
#include <string>
#include <algorithm>
#include <iomanip> // for better formatting
#include <fstream>
#include <chrono>
#include "utils.h"

#define N_MAX_THREADS_PER_BLOCK 1024
#define N_CITIES 1024

__global__ void queenAntKernel(float *choice_info, float *distances, int *tours, float *tour_lengths, int n_cities, hiprandState *states) {
    __shared__ int tabu_list[N_CITIES];
    __shared__ float probability_list[N_CITIES];

    int tid = threadIdx.x;
    int queen_id = blockIdx.x; // Each block is one queen

    hiprandState state = states[queen_id];

    if (tid < n_cities) {
        tabu_list[tid] = 1; // 1 = unvisited
        probability_list[tid] = 0.0;
    }
    __syncthreads();

    int current_city = 0;
    if (tid == 0) {
        tabu_list[current_city] = 0; // Start at city 0
        tours[queen_id * n_cities + 0] = current_city;
    }
    __syncthreads();

    float tour_len = 0.0f;

    for (int step = 1; step < n_cities; step++) {
        if (tid < n_cities) {
            probability_list[tid] = choice_info[current_city * n_cities + tid] * tabu_list[tid];
        }
        __syncthreads();

        if (tid == 0) {
            float total_prob = 0.0f;
            for (int i = 0; i < n_cities; i++) {
                total_prob += probability_list[i];
            }

            float rand_val = hiprand_uniform(&state) * total_prob;
            float cumulative = 0.0f;
            int selected_city = -1;
            for (int i = 0; i < n_cities; i++) {
                cumulative += probability_list[i];
                if (cumulative >= rand_val && rand_val > 0) {
                    selected_city = i;
                    break;
                }
            }

            if (selected_city == -1) selected_city = 0; // fallback

            tours[queen_id * n_cities + step] = selected_city;
            tour_len += distances[current_city * n_cities + selected_city];
            tabu_list[selected_city] = 0;
            current_city = selected_city;
        }
        __syncthreads();
    }

    if (tid == 0) {
        // Add return to start city
        tour_len += distances[current_city * n_cities + tours[queen_id * n_cities]];
        tour_lengths[queen_id] = tour_len;
    }

    states[queen_id] = state;
}

void queen(const std::vector<std::vector<float>>& graph, int num_iter, float alpha, float beta, float evaporate, int seed, std::string output_file) {
    std::cout << "Running QUEEN algorithm with CUDA...\n";

    auto start_total = std::chrono::high_resolution_clock::now();
    auto total_kernel = std::chrono::duration<double>::zero();
    auto total_pheromone = std::chrono::duration<double>::zero();

    int n_cities = graph.size();
    int m = n_cities; // number of queens = number of cities
    float Q = 1.0f;

    size_t matrix_size = n_cities * n_cities * sizeof(float);
    size_t array_size = m * n_cities * sizeof(int);
    size_t tour_lengths_size = m * sizeof(float);

    // Host distances matrix
    std::cout << "Host distances" << std::endl;
    std::vector<float> distances_host(n_cities * n_cities);
    for (int i = 0; i < n_cities; ++i) {
        for (int j = 0; j < n_cities; ++j) {
            distances_host[i * n_cities + j] = graph[i][j];
            std::cout << graph[i][j] << " ";
        }
        std::cout << std::endl;
    }

    // Device memory
    float *d_pheromone, *d_choice_info, *d_distances, *d_tour_lengths;
    int *d_tours;
    hiprandState* d_states;

    hipMalloc(&d_pheromone, matrix_size);
    hipMalloc(&d_choice_info, matrix_size);
    hipMalloc(&d_distances, matrix_size);
    hipMalloc(&d_tours, array_size);
    hipMalloc(&d_tour_lengths, tour_lengths_size);
    hipMalloc(&d_states, m * sizeof(hiprandState));

    hipMemcpy(d_distances, distances_host.data(), matrix_size, hipMemcpyHostToDevice);

    std::vector<float> initial_pheromone(n_cities * n_cities, 1.0f);
    hipMemcpy(d_pheromone, initial_pheromone.data(), matrix_size, hipMemcpyHostToDevice);

    int thread_queen_count = std::min(N_MAX_THREADS_PER_BLOCK, n_cities);
    int blocks_queen = std::min(N_MAX_THREADS_PER_BLOCK, n_cities);

    init_rng<<<blocks_queen, thread_queen_count>>>(d_states, seed);
    hipDeviceSynchronize();

    int all_threads_pheromone = n_cities * n_cities;
    int threads_pheromone = std::min(N_MAX_THREADS_PER_BLOCK, all_threads_pheromone);
    int blocks_pheromone = (all_threads_pheromone + threads_pheromone - 1) / threads_pheromone;

    std::vector<int> tours_host(m * n_cities);
    std::vector<float> tour_lengths_host(m);
    std::vector<float> choice_info_host(n_cities * n_cities);

    for (int iter = 0; iter < num_iter; ++iter) {
        auto start_kernel = std::chrono::high_resolution_clock::now();
        queenAntKernel<<<m, n_cities>>>(d_choice_info, d_distances, d_tours, d_tour_lengths, n_cities, d_states);
        hipDeviceSynchronize();
        auto end_kernel = std::chrono::high_resolution_clock::now();
        total_kernel += end_kernel - start_kernel;

        auto start_kernel_pheromone = std::chrono::high_resolution_clock::now();
        pheromoneUpdateKernel<<<blocks_pheromone, threads_pheromone>>>(
            alpha,
            beta,
            evaporate,
            Q,
            d_pheromone,
            d_tours,
            n_cities,
            m,
            d_choice_info,
            d_distances,
            d_tour_lengths
        );
        hipDeviceSynchronize();
        auto end_kernel_pheromone = std::chrono::high_resolution_clock::now();
        total_pheromone += end_kernel_pheromone - start_kernel_pheromone;

        // Fetch results
        // hipMemcpy(tours_host.data(), d_tours, array_size, hipMemcpyDeviceToHost);
        // hipMemcpy(tour_lengths_host.data(), d_tour_lengths, tour_lengths_size, hipMemcpyDeviceToHost);
        // hipMemcpy(choice_info_host.data(), d_choice_info, matrix_size, hipMemcpyDeviceToHost);
        // hipMemcpy(initial_pheromone.data(), d_pheromone, matrix_size, hipMemcpyDeviceToHost);

        // Print tours
        // for (int queen = 0; queen < m; ++queen) {
        //     std::cout << "Queen " << queen << " tour: ";
        //     for (int step = 0; step < n_cities; ++step) {
        //         std::cout << tours_host[queen * n_cities + step] << " ";
        //     }
        //     std::cout << " (length: " << tour_lengths_host[queen] << ")\n";
        // }

        // std::cout << "Pheromone Info Matrix:\n";
        // for (int i = 0; i < n_cities; ++i) {
        //     for (int j = 0; j < n_cities; ++j) {
        //         std::cout << std::fixed << std::setprecision(4) << initial_pheromone[i * n_cities + j] << "\t";
        //     }
        //     std::cout << "\n";
        // }

        // std::cout << "Choice Info Matrix:\n";
        // for (int i = 0; i < n_cities; ++i) {
        //     for (int j = 0; j < n_cities; ++j) {
        //         std::cout << std::fixed << std::setprecision(4) << choice_info_host[i * n_cities + j] << "\t";
        //     }
        //     std::cout << "\n";
        // }
    }

    hipMemcpy(tours_host.data(), d_tours, array_size, hipMemcpyDeviceToHost);
    hipMemcpy(tour_lengths_host.data(), d_tour_lengths, tour_lengths_size, hipMemcpyDeviceToHost);
    hipMemcpy(choice_info_host.data(), d_choice_info, matrix_size, hipMemcpyDeviceToHost);
    hipMemcpy(initial_pheromone.data(), d_pheromone, matrix_size, hipMemcpyDeviceToHost);


    float best = 1e9;
    int best_id = 0;
    for (int i = 0; i < m; ++i) {
        if (tour_lengths_host[i] < best) {
            best = tour_lengths_host[i];
            best_id = i;
        }
    }

    hipFree(d_pheromone);
    hipFree(d_choice_info);
    hipFree(d_distances);
    hipFree(d_tours);
    hipFree(d_tour_lengths);
    hipFree(d_states);

    auto end_total = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> total_duration = end_total - start_total;

    std::cout << "Total kernel time: " << total_kernel.count() << std::endl;
    std::cout << "Total pheromone update time: " << total_pheromone.count() << std::endl;
    std::cout << "Total execution time: " << total_duration.count() << std::endl;

    std::string output_path = prepare_output_path(output_file);
    std::ofstream out(output_path);
    if (!out.is_open()) {
        std::cerr << "Failed to open output file: " << output_path << std::endl;
        return;
    }

    std::cout << "\nBest tour length: " << best << std::endl;
    out << "Best tour length: " << best << std::endl;

    for (int step = 0; step < n_cities; ++step) {
        std::cout << tours_host[best_id * n_cities + step] << " ";
        out << tours_host[best_id * n_cities + step] << " ";
    }
    std::cout << std::endl;
    out << std::endl;

    out.close();
}
