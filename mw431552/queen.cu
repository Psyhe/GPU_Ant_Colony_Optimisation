#include "hip/hip_runtime.h"
#include "algorithm.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <iostream>
#include <fstream>
#include <vector>
#include <string>
#include <algorithm>
#include <iomanip> // for better formatting
#include <fstream>
#include <chrono>
#include "utils.h"

#define N_MAX_THREADS_PER_BLOCK 1024
#define N_CITIES 1024
#define N_CURRENT_CITIES 1

__global__ void queenAntKernel(float *choice_info, float *distances, int *tours, float *tour_lengths, int n_cities, hiprandState *states) {

    __shared__ int tabu[MAX_CITIES];
    __shared__ float probabilities[MAX_CITIES];
    __shared__ int current_city;
    
    int tid = threadIdx.x;
    
    if (tid >= n_cities)
        return;

    int queen_id = blockIdx.x;
    int n_threads = blockDim.x;

    int *tour = &tours[queen_id * (n_cities )];
    hiprandState localState = states[queen_id];
    
    tabu[tid] = 1; // Not visited yet
    
    __syncthreads();

    float tour_len = 0.0f;

    int start = queen_id % n_cities;
    if (tid == 0) {
        tour[0] = start;
        tabu[start] = 0; // Mark start city as visited
    }
    __syncthreads();

    current_city = start;

    for (int step = 1; step < n_cities; step++) {
        probabilities[tid] = choice_info[current_city * n_cities + tid] * tabu[tid];

        __syncthreads();

        // Thread 0 does roulette wheel selection
        double total = 0.0;
        if (tid == 0) {
            for (int i = 0; i < n_cities; i++) {
                total += probabilities[i];
            }
            double r = hiprand_uniform(&localState) * total;
            double cumulative = 0.0;
            int next_city = -1;
            for (int i = 0; i < n_cities; i++) {
                cumulative += probabilities[i];
                if (cumulative >= r) {
                    next_city = i;
                    break;
                }
            }
            if (next_city == -1) {
                // fallback
                for (int i = 0; i < n_cities; i++) {
                    if (tabu[i]) {
                        next_city = i;
                        break;
                    }
                }
            }
            tour[step] = next_city;
            tabu[next_city] = 0; // mark as visited
            tour_len += distances[current_city * n_cities + next_city];
            current_city = next_city;
        }
        __syncthreads();
    }

    if (tid == 0) {
        tour_len += distances[n_cities * queen_id + current_city];
        tour_lengths[queen_id] = tour_len;
        states[queen_id] = localState;
    }
}

void queen(const std::vector<std::vector<float>>& graph, int num_iter, float alpha, float beta, float evaporate, int seed, std::string output_file) {
        std::cout << "Running QUEEN WORKER algorithm with CUDA...\n";
    
        auto start_total = std::chrono::high_resolution_clock::now();
    
        auto total_kernel = std::chrono::duration<double>::zero();
        auto total_pheromone = std::chrono::duration<double>::zero();
    
        int n_cities = graph.size();
        int m = n_cities; // number of ants = number of cities
        float Q = 1.0f;
    
        size_t matrix_size = n_cities * n_cities * sizeof(float);
        size_t array_size = m * n_cities * sizeof(int);
        size_t tour_lengths_size = m * sizeof(float);
    
        // Host distances matrix
        std::vector<float> distances_host(n_cities * n_cities);
        for (int i = 0; i < n_cities; ++i) {
            for (int j = 0; j < n_cities; ++j) {
                distances_host[i * n_cities + j] = graph[i][j];
            }
        }
    
        // Device memory
        float *d_pheromone, *d_choice_info, *d_distances, *d_tour_lengths;
        int *d_tours;
        hiprandState* d_states;
    
        hipMalloc(&d_pheromone, matrix_size);
        hipMalloc(&d_choice_info, matrix_size);
        hipMalloc(&d_distances, matrix_size);
        hipMalloc(&d_tours, array_size);
        hipMalloc(&d_tour_lengths, tour_lengths_size);
        hipMalloc(&d_states, m * sizeof(hiprandState));
    
        hipMemcpy(d_distances, distances_host.data(), matrix_size, hipMemcpyHostToDevice);
        std::vector<float> initial_pheromone(n_cities * n_cities, 1.0f);
        hipMemcpy(d_pheromone, initial_pheromone.data(), matrix_size, hipMemcpyHostToDevice);
    
        int n_ants = n_cities;
    
        int thread_worker_count = n_cities; // one thread per city
        int blocks_worker = (n_ants); // one block per ant
    
        int all_threads_pheromone = n_ants * n_ants;
        int threads_pheromone = std::min(N_MAX_THREADS_PER_BLOCK, all_threads_pheromone);
        int blocks_pheromone = (all_threads_pheromone + threads_pheromone - 1) / threads_pheromone;
    
        init_rng<<<1, n_ants>>>(d_states, seed); // one RNG per ant
        hipDeviceSynchronize();
    
        // Host buffers to fetch data back
        std::vector<int> tours_host(m * n_cities);
        std::vector<float> choice_info_host(n_cities * n_cities);
        std::vector<float> tour_lengths_host(m);
    
        for (int iter = 0; iter < num_iter; ++iter) {
            auto start_kernel = std::chrono::high_resolution_clock::now();
    
            // CALL your queenAntKernel
            queenAntKernel<<<m, n_cities>>>(
                d_choice_info,
                d_distances,
                d_tours,
                d_tour_lengths,
                n_cities,
                d_states
            );
            hipDeviceSynchronize();
    
            auto end_kernel = std::chrono::high_resolution_clock::now();
            total_kernel += end_kernel - start_kernel;
    
            auto start_pheromone = std::chrono::high_resolution_clock::now();
            pheromoneUpdateKernel<<<blocks_pheromone, threads_pheromone>>>(
                alpha, 
                beta,
                evaporate,
                Q,
                d_pheromone,
                d_tours,
                n_cities,
                m,
                d_choice_info,
                d_distances,
                d_tour_lengths
            );
            hipDeviceSynchronize();
            auto end_pheromone = std::chrono::high_resolution_clock::now();
            total_pheromone += end_pheromone - start_pheromone;
        }
    
        hipMemcpy(tours_host.data(), d_tours, array_size, hipMemcpyDeviceToHost);
        hipMemcpy(tour_lengths_host.data(), d_tour_lengths, tour_lengths_size, hipMemcpyDeviceToHost);
        hipMemcpy(choice_info_host.data(), d_choice_info, matrix_size, hipMemcpyDeviceToHost);
        hipMemcpy(initial_pheromone.data(), d_pheromone, matrix_size, hipMemcpyDeviceToHost);
    
        float best = 1e9;
        int best_id = 0;
        for (int i = 0; i < m; ++i) {
            if (tour_lengths_host[i] < best) {
                best = tour_lengths_host[i];
                best_id = i;
            }
        }
    
        hipFree(d_pheromone);
        hipFree(d_choice_info);
        hipFree(d_distances);
        hipFree(d_tours);
        hipFree(d_tour_lengths);
        hipFree(d_states);
    
        auto end_total = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double> total_duration = end_total - start_total;
    
        std::cout << "Total kernel time: " << total_kernel.count() << std::endl;
        std::cout << "Total pheromone update time: " << total_pheromone.count() << std::endl;
        std::cout << "Total time: " << total_duration.count() << std::endl;
    
        std::string output_path = prepare_output_path(output_file);
        std::ofstream out(output_path);
    
        if (!out.is_open()) {
            std::cerr << "Failed to open output file: " << output_path << std::endl;
            return;
        }
    
        std::cout << "\nBest tour length: " << best << std::endl;
        out << "Best tour length: " << best << std::endl;
    
        for (int step = 0; step < n_cities; ++step) {
            std::cout << tours_host[best_id * n_cities + step] << " ";
            out << tours_host[best_id * n_cities + step] << " ";
        }
        std::cout << std::endl;
        out << std::endl;
    
        out.close();
    

}
