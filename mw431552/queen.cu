#include "hip/hip_runtime.h"
#include "algorithm.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <iostream>
#include <fstream>
#include <vector>
#include <string>
#include <algorithm>
#include <iomanip> // for better formatting
#include <fstream>
#include <chrono>
#include "utils.h"

#define N_MAX_THREADS_PER_BLOCK 1024
#define N_CITIES 1024
#define N_CURRENT_CITIES 1

__global__ void queenAntKernelOptimized(float *choice_info, float *distances, int *tours, float *tour_lengths, int n_cities, hiprandState *states) {

    __shared__ int tabu[N_CITIES];
    __shared__ float probabilities[N_CITIES];
    __shared__ int current_city;
    
    int tid = threadIdx.x;
    
    if (tid >= n_cities)
        return;

    int queen_id = blockIdx.x;
    int n_threads = blockDim.x;

    int *tour = &tours[queen_id * n_cities];
    hiprandState localState = states[queen_id];
    
    tabu[tid] = 1; // Not visited yet
    
    __syncthreads();

    float tour_len = 0.0f;

    int start = queen_id % n_cities;
    if (tid == 0) {
        tour[0] = start;
        tabu[start] = 0; // Mark start city as visited
    }
    __syncthreads();

    current_city = start;

    for (int step = 1; step < n_cities; step++) {
        probabilities[tid] = choice_info[current_city * n_cities + tid] * tabu[tid];
        
        __syncthreads();

        // Warp-level reduction to compute total probability
        float local_prob = probabilities[tid];
        
        // Use warp shuffle reduction
        for (int offset = 16; offset > 0; offset /= 2) {
            local_prob += __shfl_down_sync(0xffffffff, local_prob, offset);
        }

        __shared__ float total;
        if (tid % 32 == 0) { // Only one thread per warp writes
            atomicAdd(&total, local_prob);
        }
        __syncthreads();

        if (tid == 0) {
            double r = hiprand_uniform(&localState) * total;
            double cumulative = 0.0;
            int next_city = -1;
            for (int i = 0; i < n_cities; i++) {
                cumulative += probabilities[i];
                if (cumulative >= r) {
                    next_city = i;
                    break;
                }
            }
            if (next_city == -1) {
                // fallback
                for (int i = 0; i < n_cities; i++) {
                    if (tabu[i]) {
                        next_city = i;
                        break;
                    }
                }
            }
            tour[step] = next_city;
            tabu[next_city] = 0; // mark as visited
            tour_len += distances[current_city * n_cities + next_city];
            current_city = next_city;
        }
        __syncthreads();
    }

    if (tid == 0) {
        tour_len += distances[current_city * n_cities + tour[0]]; // Assuming you want a full tour
        tour_lengths[queen_id] = tour_len;
        states[queen_id] = localState;
    }
}


__global__ void queenAntKernel(float *choice_info, float *distances, int *tours, float *tour_lengths, int n_cities, hiprandState *states) {

    __shared__ int tabu[N_CITIES];
    __shared__ float probabilities[N_CITIES];
    __shared__ int current_city;
    
    int tid = threadIdx.x;
    
    if (tid >= n_cities)
        return;

    int queen_id = blockIdx.x;
    int n_threads = blockDim.x;

    int *tour = &tours[queen_id * (n_cities )];
    hiprandState localState = states[queen_id];
    
    tabu[tid] = 1; // Not visited yet
    
    __syncthreads();

    float tour_len = 0.0f;

    int start = queen_id % n_cities;
    if (tid == 0) {
        tour[0] = start;
        tabu[start] = 0; // Mark start city as visited
    }
    __syncthreads();

    current_city = start;

    for (int step = 1; step < n_cities; step++) {
        probabilities[tid] = choice_info[current_city * n_cities + tid] * tabu[tid];

        __syncthreads();

        // Thread 0 does roulette wheel selection
        double total = 0.0;
        if (tid == 0) {
            for (int i = 0; i < n_cities; i++) {
                total += probabilities[i];
            }
            double r = hiprand_uniform(&localState) * total;
            double cumulative = 0.0;
            int next_city = -1;
            for (int i = 0; i < n_cities; i++) {
                cumulative += probabilities[i];
                if (cumulative >= r) {
                    next_city = i;
                    break;
                }
            }
            if (next_city == -1) {
                // fallback
                for (int i = 0; i < n_cities; i++) {
                    if (tabu[i]) {
                        next_city = i;
                        break;
                    }
                }
            }
            tour[step] = next_city;
            tabu[next_city] = 0; // mark as visited
            tour_len += distances[current_city * n_cities + next_city];
            current_city = next_city;
        }
        __syncthreads();
    }

    if (tid == 0) {
        tour_len += distances[n_cities * queen_id + current_city];
        tour_lengths[queen_id] = tour_len;
        states[queen_id] = localState;
    }
}

void queen_old(const std::vector<std::vector<float>>& graph, int num_iter, float alpha, float beta, float evaporate, int seed, std::string output_file) {
    std::cout << "Running QUEEN WORKER algorithm with CUDA...\n";

    hipEvent_t start_total, end_total;
    hipEventCreate(&start_total);
    hipEventCreate(&end_total);
    hipEventRecord(start_total);

    float total_kernel = 0.0f;
    float total_pheromone = 0.0f;

    int n_cities = graph.size();
    int m = n_cities; // number of ants = number of cities
    float Q = 1.0f;

    size_t matrix_size = n_cities * n_cities * sizeof(float);
    size_t array_size = m * n_cities * sizeof(int);
    size_t tour_lengths_size = m * sizeof(float);

    // Host distances matrix
    std::vector<float> distances_host(n_cities * n_cities);
    for (int i = 0; i < n_cities; ++i) {
        for (int j = 0; j < n_cities; ++j) {
            distances_host[i * n_cities + j] = graph[i][j];
        }
    }

    // Device memory
    float *d_pheromone, *d_choice_info, *d_distances, *d_tour_lengths;
    int *d_tours;
    hiprandState* d_states;

    hipMalloc(&d_pheromone, matrix_size);
    hipMalloc(&d_choice_info, matrix_size);
    hipMalloc(&d_distances, matrix_size);
    hipMalloc(&d_tours, array_size);
    hipMalloc(&d_tour_lengths, tour_lengths_size);
    hipMalloc(&d_states, m * sizeof(hiprandState));

    hipMemcpy(d_distances, distances_host.data(), matrix_size, hipMemcpyHostToDevice);
    std::vector<float> initial_pheromone(n_cities * n_cities, 1.0f);
    hipMemcpy(d_pheromone, initial_pheromone.data(), matrix_size, hipMemcpyHostToDevice);

    int n_ants = n_cities;

    int thread_worker_count = n_cities; // one thread per city
    int blocks_worker = (n_ants); // one block per ant

    int all_threads_pheromone = n_ants * n_ants;
    int threads_pheromone = std::min(N_MAX_THREADS_PER_BLOCK, all_threads_pheromone);
    int blocks_pheromone = (all_threads_pheromone + threads_pheromone - 1) / threads_pheromone;

    init_rng<<<1, n_ants>>>(d_states, seed); // one RNG per ant
    hipDeviceSynchronize();

    // Host buffers to fetch data back
    std::vector<int> tours_host(m * n_cities);
    std::vector<float> choice_info_host(n_cities * n_cities);
    std::vector<float> tour_lengths_host(m);

    hipEvent_t start_kernel, end_kernel;
    hipEvent_t start_pheromone, end_pheromone;
    hipEventCreate(&start_kernel);
    hipEventCreate(&end_kernel);
    hipEventCreate(&start_pheromone);
    hipEventCreate(&end_pheromone);

    for (int iter = 0; iter < num_iter; ++iter) {
        hipEventRecord(start_kernel);

        queenAntKernel<<<m, n_cities>>>(
            d_choice_info,
            d_distances,
            d_tours,
            d_tour_lengths,
            n_cities,
            d_states
        );
        hipDeviceSynchronize();

        hipEventRecord(end_kernel);
        hipEventSynchronize(end_kernel);
        float kernel_ms = 0.0f;
        hipEventElapsedTime(&kernel_ms, start_kernel, end_kernel);
        total_kernel += kernel_ms;

        hipEventRecord(start_pheromone);

        pheromoneUpdateKernel<<<blocks_pheromone, threads_pheromone>>>(
            alpha,
            beta,
            evaporate,
            Q,
            d_pheromone,
            d_tours,
            n_cities,
            m,
            d_choice_info,
            d_distances,
            d_tour_lengths
        );
        hipDeviceSynchronize();

        hipEventRecord(end_pheromone);
        hipEventSynchronize(end_pheromone);
        float pheromone_ms = 0.0f;
        hipEventElapsedTime(&pheromone_ms, start_pheromone, end_pheromone);
        total_pheromone += pheromone_ms;
    }

    hipMemcpy(tours_host.data(), d_tours, array_size, hipMemcpyDeviceToHost);
    hipMemcpy(tour_lengths_host.data(), d_tour_lengths, tour_lengths_size, hipMemcpyDeviceToHost);
    hipMemcpy(choice_info_host.data(), d_choice_info, matrix_size, hipMemcpyDeviceToHost);
    hipMemcpy(initial_pheromone.data(), d_pheromone, matrix_size, hipMemcpyDeviceToHost);

    float best = 1e9;
    int best_id = 0;
    for (int i = 0; i < m; ++i) {
        if (tour_lengths_host[i] < best) {
            best = tour_lengths_host[i];
            best_id = i;
        }
    }

    hipFree(d_pheromone);
    hipFree(d_choice_info);
    hipFree(d_distances);
    hipFree(d_tours);
    hipFree(d_tour_lengths);
    hipFree(d_states);

    hipEventRecord(end_total);
    hipEventSynchronize(end_total);
    float total_time_ms = 0.0f;
    hipEventElapsedTime(&total_time_ms, start_total, end_total);

    hipEventDestroy(start_kernel);
    hipEventDestroy(end_kernel);
    hipEventDestroy(start_pheromone);
    hipEventDestroy(end_pheromone);
    hipEventDestroy(start_total);
    hipEventDestroy(end_total);

    std::cout << "Total kernel time: " << total_kernel / 1000.0f << " seconds" << std::endl;
    std::cout << "Total pheromone update time: " << total_pheromone / 1000.0f << " seconds" << std::endl;
    std::cout << "Average kernel time: " << total_kernel / num_iter << " ms" << std::endl;
    std::cout << "Average pheromone kernel time: " << total_pheromone / num_iter << " ms" << std::endl;

    std::cout << "Total time: " << total_time_ms / 1000.0f << " seconds" << std::endl;

    std::string output_path = prepare_output_path(output_file);
    std::ofstream out(output_path);

    if (!out.is_open()) {
        std::cerr << "Failed to open output file: " << output_path << std::endl;
        return;
    }

    std::cout << "\nBest tour length: " << best << std::endl;
    out << "Best tour length: " << best << std::endl;

    for (int step = 0; step < n_cities; ++step) {
        std::cout << tours_host[best_id * n_cities + step] << " ";
        out << tours_host[best_id * n_cities + step] + 1 << " ";
    }
    std::cout << std::endl;
    out << std::endl;

    out.close();
}

void queen(const std::vector<std::vector<float>>& graph, int num_iter, float alpha, float beta, float evaporate, int seed, std::string output_file) {
    std::cout << "Running QUEEN WORKER algorithm with CUDA + Graphs...\n";

    hipEvent_t start_total, end_total;
    hipEventCreate(&start_total);
    hipEventCreate(&end_total);
    hipEventRecord(start_total);

    float total_kernel = 0.0f;
    float total_pheromone = 0.0f;

    int n_cities = graph.size();
    int m = n_cities;
    float Q = 1.0f;

    size_t matrix_size = n_cities * n_cities * sizeof(float);
    size_t array_size = m * n_cities * sizeof(int);
    size_t tour_lengths_size = m * sizeof(float);

    std::vector<float> distances_host(n_cities * n_cities);
    for (int i = 0; i < n_cities; ++i) {
        for (int j = 0; j < n_cities; ++j) {
            distances_host[i * n_cities + j] = graph[i][j];
        }
    }

    // Device memory
    float *d_pheromone, *d_choice_info, *d_distances, *d_tour_lengths;
    int *d_tours;
    hiprandState* d_states;

    hipMalloc(&d_pheromone, matrix_size);
    hipMalloc(&d_choice_info, matrix_size);
    hipMalloc(&d_distances, matrix_size);
    hipMalloc(&d_tours, array_size);
    hipMalloc(&d_tour_lengths, tour_lengths_size);
    hipMalloc(&d_states, m * sizeof(hiprandState));

    hipMemcpy(d_distances, distances_host.data(), matrix_size, hipMemcpyHostToDevice);
    std::vector<float> initial_pheromone(n_cities * n_cities, 1.0f);
    hipMemcpy(d_pheromone, initial_pheromone.data(), matrix_size, hipMemcpyHostToDevice);

    int n_ants = n_cities;

    int thread_worker_count = n_cities; // one thread per city
    int blocks_worker = (n_ants); // one block per ant

    int all_threads_pheromone = n_ants * n_ants;
    int threads_pheromone = std::min(N_MAX_THREADS_PER_BLOCK, all_threads_pheromone);
    int blocks_pheromone = (all_threads_pheromone + threads_pheromone - 1) / threads_pheromone;

    init_rng<<<1, n_ants>>>(d_states, seed);
    hipDeviceSynchronize();

    // Host buffers
    std::vector<int> tours_host(m * n_cities);
    std::vector<float> choice_info_host(n_cities * n_cities);
    std::vector<float> tour_lengths_host(m);

    hipStream_t stream;
    hipStreamCreate(&stream);

    hipGraph_t graph_capture;
    hipGraphExec_t graph_exec;

    // Start capturing the kernel and pheromone updates into a graph
    hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);

    queenAntKernel<<<m, n_cities, 0, stream>>>(
        d_choice_info,
        d_distances,
        d_tours,
        d_tour_lengths,
        n_cities,
        d_states
    );

    pheromoneUpdateKernel<<<blocks_pheromone, threads_pheromone, 0, stream>>>(
        alpha,
        beta,
        evaporate,
        Q,
        d_pheromone,
        d_tours,
        n_cities,
        m,
        d_choice_info,
        d_distances,
        d_tour_lengths
    );

    hipStreamEndCapture(stream, &graph_capture);
    hipGraphInstantiate(&graph_exec, graph_capture, NULL, NULL, 0);

    runGraphIterations(graph_exec, stream, num_iter, total_kernel);


    hipMemcpy(tours_host.data(), d_tours, array_size, hipMemcpyDeviceToHost);
    hipMemcpy(tour_lengths_host.data(), d_tour_lengths, tour_lengths_size, hipMemcpyDeviceToHost);
    hipMemcpy(choice_info_host.data(), d_choice_info, matrix_size, hipMemcpyDeviceToHost);
    hipMemcpy(initial_pheromone.data(), d_pheromone, matrix_size, hipMemcpyDeviceToHost);

    float best = 1e9;
    int best_id = 0;
    for (int i = 0; i < m; ++i) {
        if (tour_lengths_host[i] < best) {
            best = tour_lengths_host[i];
            best_id = i;
        }
    }

    // Cleanup
    hipFree(d_pheromone);
    hipFree(d_choice_info);
    hipFree(d_distances);
    hipFree(d_tours);
    hipFree(d_tour_lengths);
    hipFree(d_states);

    hipGraphDestroy(graph_capture);
    hipGraphExecDestroy(graph_exec);
    hipStreamDestroy(stream);

    hipEventRecord(end_total);
    hipEventSynchronize(end_total);

    float total_time_ms = 0.0f;
    hipEventElapsedTime(&total_time_ms, start_total, end_total);

    hipEventDestroy(start_total);
    hipEventDestroy(end_total);

    generate_output(total_kernel, num_iter, total_time_ms, output_file, tours_host, best_id, best, n_cities);
}
